#include "hip/hip_runtime.h"
/*
    *   Arturo Alam Tellez Villagomez
    *   A01205569
    *   Matrix Multiplication
    *   Multiprocessors
    *   Final Project
*/

#include <stdio.h>
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include "../utils/cheader.h"

#define BLOCK_SIZE 16

__global__ void matMult(int *a, int *b, int *c, int filasA, int columnasB, int k) {
  // This somehow does the job
  float temp = 0;
  int i;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // If the tid is greater than the size of the result matrix, return
  if(row > filasA || col > columnasB) return;
  for (i = 0; i < k; ++i)
    temp += (a[row * k + i]) * (b[i * columnasB + col]);
  c[row * columnasB + col] = temp;

}

int main(){
  // Declare variables
  int *a, *b, *c;
  int *dev_a;
  int *dev_b;
  int *dev_c;
  double ms;
  // Columns and rows to use
  int filasA;
  int columnasA;
  int filasB;
  int columnasB;
  // Get values for rows and columns
  printf ("Rows of matrix A: ");
  scanf ("%i",&filasA);
  printf ("Columns of matrix A: ");
  scanf ("%i",&columnasA);
  filasB = columnasA;
  printf ("Columns of matrix B: ");
  scanf ("%i",&columnasB);
  /*
  filasA = 5;
  columnasA = 5;
  filasB = columnasA;
  columnasB = 5;
  */
  // Define sizes to store in both host and device
  int sizeA = filasA * columnasA * sizeof(int);
  int sizeB = filasB * columnasB * sizeof(int);
  int sizeC = filasA * columnasB * sizeof(int);

  // define matrixes
  a = (int*)malloc(sizeA);
  b = (int*)malloc(sizeB);
  c = (int*)malloc(sizeC);
  // Fill matrix A
  for(int i = 0; i < filasA; i++)
    for(int j = 0; j < columnasA; j++)
      a[i*columnasA + j] = (int)((rand() % 100) + 1);
  //Fill matrix B
  for(int i = 0; i < filasB; i++)
    for(int j = 0; j < columnasB; j++)
      b[i*columnasB + j] = (int)((rand() % 100) + 1);
  // Allocate values in device
  hipMalloc((void**) &dev_a, sizeA);
  hipMalloc((void**) &dev_b, sizeB);
  hipMalloc((void**) &dev_c, sizeC);
  // Copy the values from matrix A and B in host to the GPU
  hipMemcpy(dev_a, a, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeB, hipMemcpyHostToDevice);
  // Define the grid that is going to have onw thread per matrix location of the result
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((columnasB + dimBlock.x - 1) / dimBlock.x, (filasA + dimBlock.y - 1) / dimBlock.y);
  // Call function in GPU
  ms = 0;
  start_timer();
  matMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, filasA, columnasB, columnasA);
  ms = stop_timer();
  // Get the value from GPU to Host so we can print it
  hipMemcpy(c, dev_c, sizeC, hipMemcpyDeviceToHost);
  //Print the three matrixes
  printf("Matrix A \n");
  for(int i = 0; i < filasA; i++){
    for(int j = 0; j < columnasA; j++){
      printf("%i ", a[i*columnasA + j]);
    }
    printf("\n");
  }
  printf("\n");
  printf("Matrix B \n");
  for(int i = 0; i < filasB; i++){
    for(int j = 0; j < columnasB; j++){
      printf("%i ", b[i*columnasB + j]);
    }
    printf("\n");
  }
  printf("\n");
  printf("Matrix C (Result) \n");
  for(int i = 0; i < filasA; i++){
    for(int j = 0; j < columnasB; j++){
      printf("%i \t", c[i*columnasB + j]);
    }
    printf("\n");
  }
  printf("\n");

  printf("Time = %lf\n", ms);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  free(a);
  free(b);
  free(c);
}

